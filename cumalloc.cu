#include <stdio.h>
#include "hip/hip_runtime.h"
#include <iostream>
#include <unistd.h>
using namespace std;

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

int main(int argc, char* argv[])
{
  //allocating and initializing device buffers
  int size = 10;
  int sleep_duration = 99999999;
  cout << "please input size (MB)\n" << endl;
  // cin >> size;
  cout << "default with " << size << " MB\n" << endl;
  int malloc_size = size *1024 *1024 /4;
  printf("receiving value %d, size with %d (floats)\n", size, malloc_size);
  float* sendbuff = (float*)malloc(sizeof(float*) );

  CUDACHECK(hipMalloc(&sendbuff, malloc_size * sizeof(float)));
  // cout << "press any key to end" << endl;
  // char tmp[50];
  // cin >> tmp;
  sleep(sleep_duration);
  return 0;
}
